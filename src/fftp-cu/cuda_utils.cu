#include "cuda_utils.hcu"
/***************************************************************
! cuda_utils.cu
!
! CUDA interface routines. Generally, contained here are those
! methods that require at least some CUDA. Those that are
! mainly interfaces to the CUDA RTL are contained in 'cuda_bindings.f90'
!
! 2011 Duane Rosenberg & Pablo D. Mininni
!      National Center for Atmospheric Research
!      e-mail: mininni@df.uba.ar 
!
***************************************************************/
#include <hipfft/hipfft.h>

extern "C" {

/* Memcpy methods: */
hipError_t cudaMemcpyHost2Dev( void *devdst, const void *hostsrc, size_t count)
{
  hipError_t iret;
  iret = hipMemcpy( devdst, hostsrc, count, hipMemcpyHostToDevice ) ;
  return iret;
}

hipError_t cudaMemcpyDev2Host( void *hostdst, const void *devsrc, size_t count)
{
  hipError_t iret;
  iret = hipMemcpy( hostdst , devsrc, count, hipMemcpyDeviceToHost ); 
  return iret;
}

hipError_t cudaMemcpyAsyncHost2Dev( void *devdst, const void *hostsrc, size_t count, hipStream_t *stream)
{
  hipError_t iret;
  iret = hipMemcpyAsync( devdst, hostsrc, count, hipMemcpyHostToDevice, *stream );
  return iret;
}

hipError_t cudaMemcpyAsyncDev2Host( void *hostdst, const void *devsrc, size_t count, hipStream_t *stream)
{
  hipError_t iret;
  iret = hipMemcpyAsync( hostdst, devsrc, count, hipMemcpyDeviceToHost, *stream ); 
  return iret;
}

hipError_t cudaMemcpyAsyncOffHost2Dev( void *devdst, size_t byteoffdev, const void *hostsrc, size_t byteoffhost, size_t count, hipStream_t *stream)
{
  hipError_t iret;
  iret = hipMemcpyAsync( (char *) devdst + byteoffdev, (char *) hostsrc + byteoffhost, count, hipMemcpyHostToDevice, *stream );
  return iret;
}

hipError_t cudaMemcpyAsyncOffDev2Host( void *hostdst, size_t byteoffhost, const void *devsrc, size_t byteoffdev, size_t count, hipStream_t *stream)
{
  hipError_t iret;
  iret = hipMemcpyAsync( (char *) hostdst + byteoffhost, (char *) devsrc + byteoffdev, count, hipMemcpyDeviceToHost, *stream ); 
  return iret;
}

hipfftResult cufftPlanManyNULL( hipfftHandle *plan, int rank, int *n, int *inembed, int istride, int idist, int *onembed, int ostride, int odist, hipfftType type, int batch)
{
  hipfftResult iret;
  iret = hipfftPlanMany(plan, rank, n, NULL, istride, idist, NULL, ostride, odist, type, batch);
  return iret;
}

/* Stream methods: */
hipError_t ptr_cudaStreamCreate( hipStream_t **stream)
{
  *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
  return hipStreamCreate( *stream );
}

hipError_t f_cudaStreamSynchronize( hipStream_t *stream)
{
  hipError_t iret;
  iret = hipStreamSynchronize( *stream );
  return iret;
}

hipfftResult f_cufftSetStream( hipfftHandle plan, hipStream_t *stream)
{
  hipfftResult iret;
  iret = hipfftSetStream( plan, *stream );
  return iret;
}

/* Interfaces for cuFFT with offsets: */

hipfftResult cufftExecOffC2R( hipfftHandle plan, void *datain, size_t byteoffin, void *dataout, size_t byteoffout)
{
  hipfftResult iret;
  char* ptrin  = (char *) datain  + byteoffin;
  char* ptrout = (char *) dataout + byteoffout;
  iret = hipfftExecC2R( plan, (hipfftComplex *) ptrin, (hipfftReal *) ptrout );
  return iret;
}

hipfftResult cufftExecOffR2C( hipfftHandle plan, void *datain, size_t byteoffin, void *dataout, size_t byteoffout)
{
  hipfftResult iret;
  char* ptrin  = (char *) datain  + byteoffin;
  char* ptrout = (char *) dataout + byteoffout;

  iret = hipfftExecR2C( plan, (hipfftReal *) ptrin, (hipfftComplex *) ptrout );
  return iret;
}

hipfftResult cufftExecOffC2C( hipfftHandle plan, void *datain, size_t byteoffin, void *dataout, size_t byteoffout, int dir)
{
  hipfftResult iret;
  char* ptrin  = (char *) datain  + byteoffin;
  char* ptrout = (char *) dataout + byteoffout;
  iret = hipfftExecC2C( plan, (hipfftComplex *) ptrin, (hipfftComplex *) ptrout, dir );
  return iret;
}

hipfftResult cufftExecOffZ2D( hipfftHandle plan, void *datain, size_t byteoffin, void *dataout, size_t byteoffout)
{
  hipfftResult iret;
  char* ptrin  = (char *) datain  + byteoffin;
  char* ptrout = (char *) dataout + byteoffout;
  iret = hipfftExecZ2D( plan, (hipfftDoubleComplex *) ptrin, (hipfftDoubleReal *) ptrout );
  return iret;
}

hipfftResult cufftExecOffD2Z( hipfftHandle plan, void *datain, size_t byteoffin, void *dataout, size_t byteoffout)
{
  hipfftResult iret;
  char* ptrin  = (char *) datain  + byteoffin;
  char* ptrout = (char *) dataout + byteoffout;
  iret = hipfftExecD2Z( plan, (hipfftDoubleReal *) ptrin, (hipfftDoubleComplex *) ptrout );
  return iret;
}

hipfftResult cufftExecOffZ2Z( hipfftHandle plan, void *datain, size_t byteoffin, void *dataout, size_t byteoffout, int dir)
{
  hipfftResult iret;
  char* ptrin  = (char *) datain  + byteoffin;
  char* ptrout = (char *) dataout + byteoffout;
  iret = hipfftExecZ2Z( plan, (hipfftDoubleComplex *) ptrin, (hipfftDoubleComplex *) ptrout, dir );
  return iret;
}

} /* end, extern "C" interface */


